#include "hip/hip_runtime.h"
/**
 * @author Federico Busato                                                  <br>
 *         Univerity of Verona, Dept. of Computer Science                   <br>
 *         federico.busato@univr.it
 * @date September, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 Hornet. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 */
#include "Static/ShortestPath/SSSP.cuh"
#include <GraphIO/GraphWeight.hpp>
#include <GraphIO/BellmanFord.hpp>

namespace hornets_nest {

const weight_t INF = std::numeric_limits<weight_t>::max();

//------------------------------------------------------------------------------
///////////////
// OPERATORS //
///////////////

struct SSSPOperator {
    weight_t*            d_distances;
    TwoLevelQueue<vid_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge) {
        auto       src = vertex.id();
        auto       dst = edge.dst_id();
        auto    weight = edge.weight();
        auto tentative = d_distances[src] + weight;
        if (tentative < d_distances[dst]) {
            d_distances[dst] = tentative;
            queue.insert(dst);
        }
    }
};
//------------------------------------------------------------------------------
/////////////////
// SSSP //
/////////////////

SSSP::SSSP(HornetGraph& hornet) : StaticAlgorithm(hornet),
                                queue(hornet),
                                load_balacing(hornet) {
    gpu::allocate(d_distances, hornet.nV());
    reset();
}

SSSP::~SSSP() {
    gpu::free(d_distances);
}

void SSSP::reset() {
    queue.clear();
    auto distances = d_distances;
    forAllnumV(hornet, [=] __device__ (int i){ distances[i] = INF; } );
}

void SSSP::set_parameters(vid_t source) {
    sssp_source = source;
    queue.insert(sssp_source);
    host::copyToDevice(weight_t(0), d_distances + sssp_source);
}

void SSSP::run() {
    while (queue.size() > 0) {
        forAllEdges(hornet, queue, SSSPOperator { d_distances, queue },
                    load_balacing);
        queue.swap();
    }
}

void SSSP::release() {
    gpu::free(d_distances);
    d_distances = nullptr;
}

bool SSSP::validate() {
    using namespace graph;
    GraphWeight<vid_t, eoff_t, weight_t>
        graph(hornet.csr_offsets(), hornet.nV(),
              hornet.csr_edges(), hornet.nE(), hornet.edge_field<1>());
    BellmanFord<vid_t, eoff_t, weight_t> sssp(graph);
    sssp.run(sssp_source);

    auto h_distances = sssp.result();
    return gpu::equal(h_distances, h_distances + graph.nV(), d_distances);
}

} // namespace hornets_nest
