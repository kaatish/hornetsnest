/**
 * @brief Triangle test program
 * @file
 */

#include "HornetAlg.hpp"
#include "Core/LoadBalancing/VertexBased.cuh"
#include "Core/LoadBalancing/ScanBased.cuh"
#include "Core/LoadBalancing/BinarySearch.cuh"
#include <Core/GPUCsr/Csr.cuh>
#include <Core/GPUHornet/Hornet.cuh>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include "Static/TriangleCounting/triangle2.cuh"

using namespace timer;
using namespace hornets_nest;

using HornetGraph = gpu::Hornet<EMPTY, EMPTY>;


// CPU Version - assume sorted index lists.
int hostSingleIntersection (const vid_t ai, const degree_t alen, const vid_t * a,
                            const vid_t bi, const degree_t blen, const vid_t * b){

     int32_t out = 0;


    if (!alen || !blen || a[alen-1] < b[0] || b[blen-1] < a[0])
    return 0;

    const vid_t *aptr=a, *aend=a+alen;
    const vid_t *bptr=b, *bend=b+blen;

    while(aptr< aend && bptr<bend){
        if(*aptr==*bptr){
            aptr++, bptr++, out++;
        }
        else if(*aptr<*bptr){
            aptr++;
        }
        else {
            bptr++;
        }
      }

    return out;
}

void hostCountTriangles (const vid_t nv, const vid_t ne, const eoff_t * off,
    const vid_t * ind, int64_t* allTriangles)
{
    int64_t sum=0;
    for (vid_t src = 0; src < nv; src++)
    {
        degree_t srcLen=off[src+1]-off[src];
        for(int iter=off[src]; iter<off[src+1]; iter++)
        {
            vid_t dest=ind[iter];
            degree_t destLen=off[dest+1]-off[dest];
            int64_t tris= hostSingleIntersection (src, srcLen, ind+off[src],
                                                    dest, destLen, ind+off[dest]);
            sum+=tris;
        }
    }
    *allTriangles=sum;
}


int main(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
    TriangleCounting2 tc(hornet_graph);
    tc.init();
    Timer<DEVICE> TM(5);
    hipProfilerStart();
    TM.start();

    tc.run();

    TM.stop();
    hipProfilerStop();
    TM.print("Computation time:");

    triangle_t deviceTriangleCount = tc.countTriangles();
    std::cout << "Device triangles: " << deviceTriangleCount << "\n";

    return 0;
}
