/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

int main(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);


    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
            graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);


    BfsTopDown2 bfs_top_down(hornet_graph);

    vid_t root = graph.max_out_degree_id();
    if (argc==3)
        root = atoi(argv[2]);

    bfs_top_down.set_parameters(root);

    Timer<DEVICE> TM;
    hipProfilerStart();
    TM.start();

    bfs_top_down.run();

    TM.stop();
    hipProfilerStop();
    TM.print("TopDown2");

    auto is_correct = bfs_top_down.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return !is_correct;
}
