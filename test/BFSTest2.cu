/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include <GraphIO/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

int main(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv);
    //graph.print();

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.out_offsets_ptr(),
                           graph.out_edges_ptr());

    HornetGraph hornet_graph(hornet_init);
    //hornet_graph.print();

    BfsTopDown2 bfs_top_down(hornet_graph);

    bfs_top_down.set_parameters(graph.max_out_degree_id());

    Timer<DEVICE> TM;
    hipProfilerStart();
    TM.start();

    bfs_top_down.run();

    TM.stop();
    hipProfilerStop();
    TM.print("TopDown2");

    auto is_correct = bfs_top_down.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return !is_correct;
}
